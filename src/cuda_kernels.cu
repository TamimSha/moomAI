
#include <hip/hip_runtime.h>
__global__ void cuda_GetImgDiff(unsigned char *dest, unsigned char *a, unsigned char *b, int res) {
    int x = 3*threadIdx.x + 3*(blockIdx.x * blockDim.x);
    int y = (3 * res)*threadIdx.y + (3 * res)*(blockIdx.y * blockDim.y);
    int z = threadIdx.z;
    int i = (x + y + z);
    if(a[i] >= b[i]){
        dest[i] = a[i] - b[i];
    }
    else{
        dest[i] = b[i] - a[i];
    }
}

__global__ void cuda_SumPixels(float *d_in, float *d_out) {
        int thId = threadIdx.x;
        int id = threadIdx.x + blockDim.x * blockIdx.x;

        for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1)
        {
            if(thId < s)
            {
                d_in[id] += d_in[id + s];
            }
            __syncthreads();
        }

        if(thId == 0)
        {
            d_out[blockIdx.x] = d_in[id];
        }
}

__global__ void cuda_ByteToFloat(float *f, unsigned char *b) {
      int x = threadIdx.x + blockDim.x * blockIdx.x;
      f[x] = (float)b[x];
}